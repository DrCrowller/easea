#include "hip/hip_runtime.h"
#define NUMTHREAD2 128
#define MAX_STACK 50
#define LOGNUMTHREAD2 7

#define NUMTHREAD 32
#define LOGNUMTHREAD 5

#define HIT_LEVEL  0.01f
#define PROBABLY_ZERO  1.11E-15f
#define BIG_NUMBER 1.0E15f


struct gpuArg{
  int threadId;
  sem_t sem_in;
  sem_t sem_out;

  float* progs_k;
  float* results_k;
  int* indexes_k;
  int* hits_k;
  float* inputs_k;
  float* outputs_k;

  int index_st;
  int index_end;

  int indiv_st;
  int indiv_end;
};

struct gpuArg* gpuArgs;
bool freeGPU = false;
int sh_pop_size = 0;
int sh_length = 0;




__global__ static void 
fastEvaluatePostFixIndividuals_32_mgpu( const float * k_progs,
					const int maxprogssize,
					const int popsize,
					const float * k_inputs,
					const float * k_outputs,
					const int trainingSetSize,
					float * k_results,
					int *k_hits,
					
					const int indivPerBlock,
					const int* indexes,
					const int start_index,
					const int gpu_id
					){

  extern __shared__ float scratch[];
  float* tmpresult = scratch+(threadIdx.y*NUMTHREAD);
  float* tmphits = scratch+(indivPerBlock*NUMTHREAD)+(threadIdx.y*NUMTHREAD);
  /* __shared__ float tmpresult[NUMTHREAD]; */
  /* __shared__ float tmphits[NUMTHREAD]; */

  const int tid = threadIdx.x; //0 to NUM_THREADS-1
  const int bid = blockIdx.x+threadIdx.y*(popsize/indivPerBlock)+blockIdx.y*gridDim.x; // 0 to NUM_BLOCKS-1

  int index;   // index of the prog processed by the block 
  float sum = 0.0;
  int hits = 0 ; // hits number

  float currentOutput;
  float result;
  int start_prog;
  int codop;
  float stack[MAX_STACK];
  int  sp, var_id;
  float op1, op2;

  index = bid; // one program per block => block ID = program number

  if (index >= popsize){ // idle block (should never occur)
    return;
  }
  if (indexes[index] == -1.0) // already evaluated
    return;


  // Here, it's a busy thread

  sum = 0.0;
  hits = 0 ; // hits number

  

  // Loop on training cases, per cluster of 32 cases (= number of thread)
  // (even if there are only 8 stream processors, we must spawn at least 32 threads) 
  // We loop from 0 to upper bound INCLUDED in case trainingSetSize is not 
  // a multiple of NUMTHREAD
  for (int i=0; i < ((trainingSetSize-1)>>LOGNUMTHREAD)+1; i++) {

    int fc_id = i*NUMTHREAD+tid;
    // are we on a busy thread?
    if ( fc_id >= trainingSetSize) // no!
      continue;

    currentOutput = k_inputs[(1+fc_id)*DRONE_VAR_LEN+DRONE_FCT_ID];
    const float* currentInputs = k_inputs+(fc_id)*DRONE_VAR_LEN;
    start_prog = indexes[index]-start_index; // index of first codop
    codop =  k_progs[start_prog++];
    
    sp = 0; // stack and stack pointer
    
    while (codop != OP_RETURN){
      switch(codop){
	case OP_VAR : 
	  var_id = k_progs[start_prog++];
	  stack[sp++] = currentInputs[var_id];
	  break;      
      case OP_ERC: stack[sp++] = k_progs[start_prog++]; break;
      case OP_MUL :
	op1 = stack[--sp]; op2 = stack[sp-1];
	stack[sp-1] = op1*op2; break;
      case OP_ADD :
	op1 = stack[--sp]; op2 = stack[sp-1];
	stack[sp-1] = op1+op2; break;
      case OP_SUB :
	op1 = stack[--sp]; op2 = stack[sp-1];
	stack[sp-1] = op2 - op1; break;
      case OP_DIV :
	op2 = stack[--sp]; op1 = stack[sp-1];
	if (op2 == 0.0) stack[sp-1] = DIV_ERR_VALUE;
	else stack[sp-1] = op1/op2;
	break;
      case OP_SIN : stack[sp-1] = sinf(stack[sp-1]); break;
      case OP_COS : stack[sp-1] = cosf(stack[sp-1]); break;
      }
      // get next codop
      codop =  k_progs[start_prog++];
    } // codop interpret loop

    result = fabsf(stack[0] - currentOutput);
    
    if (!(result < BIG_NUMBER))
      result = BIG_NUMBER;
    else if (result < PROBABLY_ZERO)
      result = 0.0;
    
    if (result <= HIT_LEVEL)
      hits++;
    
    sum += result; // sum raw error on all training cases
    
  } // LOOP ON TRAINING CASES

  // gather results from all threads => we need to synchronize (not sure, take a look to next comment)
  tmpresult[tid] = sum;
  tmphits[tid] = hits;

  //__syncthreads(); // this is useless, because warps are synchronized by nature

  if( tid == 0 ){
    for (int i = 1; i < NUMTHREAD; i++) {
      tmpresult[0] += tmpresult[i];
      tmphits[0] += tmphits[i];
    }    
    k_results[index] = tmpresult[0];
    k_hits[index] = tmphits[0];
    //printf("tid.y = %d k_results %d = %f\n",threadIdx.y,index,k_results[index]);
  }  
}




__global__ static void 
EvaluatePostFixIndividuals_128_mgpu(const float * k_progs,
				    const int maxprogssize,
				    const int popsize,
				    const float * k_inputs,
				    const float * k_outputs,
				    const int trainingSetSize,
				    float * k_results,
				    int *k_hits,
				    int* k_indexes,
				    int start_index,
				    int gpu_id
			       )
{
  __shared__ float tmpresult[NUMTHREAD2];
  __shared__ float tmphits[NUMTHREAD2];
  
  const int tid = threadIdx.x; //0 to NUM_THREADS-1
  const int bid = blockIdx.x; // 0 to NUM_BLOCKS-1

  
  int index;   // index of the prog processed by the block 
  float sum = 0.0;
  int hits = 0 ; // hits number

  float currentOutput;
  float result;
  int start_prog;
  int codop;
  float stack[MAX_STACK];
  int  sp, var_id;
  float op1, op2;

  index = bid; // one program per block => block ID = program number
 
  if (index >= popsize) // idle block (should never occur)
    return;
  if (k_progs[index] == -1.0) // already evaluated
    return;

  // Here, it's a busy thread

  sum = 0.0;
  hits = 0 ; // hits number
  
  // Loop on training cases, per cluster of 32 cases (= number of thread)
  // (even if there are only 8 stream processors, we must spawn at least 32 threads) 
  // We loop from 0 to upper bound INCLUDED in case trainingSetSize is not 
  // a multiple of NUMTHREAD
  for (int i=0; i < ((trainingSetSize-1)>>LOGNUMTHREAD2)+1; i++) {
    int fc_id = i*NUMTHREAD2+tid;
    // are we on a busy thread?
    if (fc_id >= trainingSetSize) // no!
      continue;
    
    currentOutput = k_inputs[(1+fc_id)*DRONE_VAR_LEN+DRONE_FCT_ID];
    start_prog = k_indexes[index]-start_index; // index of first codop
    codop =  k_progs[start_prog++];
    
    const float* currentInputs = k_inputs+(fc_id)*DRONE_VAR_LEN;
    sp = 0; // stack and stack pointer
    
    while (codop != OP_RETURN){
      switch(codop)
	{
	case OP_VAR : 
	  var_id = k_progs[start_prog++];
	  stack[sp++] = currentInputs[var_id];
	  break;
	case OP_ERC: stack[sp++] = k_progs[start_prog++]; break;
	case OP_MUL :
	  op1 = stack[--sp]; op2 = stack[sp-1];
	  stack[sp-1] = op1*op2; break;
	case OP_ADD :
	  op1 = stack[--sp]; op2 = stack[sp-1];
	  stack[sp-1] = op1+op2; break;
	case OP_SUB :
	  op1 = stack[--sp]; op2 = stack[sp-1];
	  stack[sp-1] = op2 - op1; break;
	case OP_DIV :
	  op2 = stack[--sp]; op1 = stack[sp-1];
	  if (op2 == 0.0) stack[sp-1] = DIV_ERR_VALUE;
	  else stack[sp-1] = op1/op2;
	  break;
	case OP_POW :
	  op2 = stack[--sp]; op1 = stack[sp-1];
	  stack[sp-1] = powf(op1,op2);
	  break;
        case OP_SIN : stack[sp-1] = sinf(stack[sp-1]); break;
        case OP_COS : stack[sp-1] = cosf(stack[sp-1]); break;
	}
      // get next codop
      codop =  k_progs[start_prog++];
    } // codop interpret loop
    result = fabsf(stack[0] - currentOutput);
    
    if (!(result < BIG_NUMBER)) result = BIG_NUMBER;
    else if (result < PROBABLY_ZERO) result = 0.0;

    if (result <= HIT_LEVEL) hits++;
    
    sum += result; // sum raw error on all training cases
    
  } // LOOP ON TRAINING CASES
  
  // gather results from all threads => we need to synchronize
  tmpresult[tid] = sum;
  tmphits[tid] = hits;
  __syncthreads();

  if (tid == 0) {
    for (int i = 1; i < NUMTHREAD2; i++) {
      tmpresult[0] += tmpresult[i];
      tmphits[0] += tmphits[i];
    }    
    k_results[index] = tmpresult[0];
    k_hits[index] = tmphits[0];
    //printf("g %d %d %f\n",gpu_id,bid,k_results[index]);
    //fflush(stdout);
  }  
  // here results and hits have been stored in their respective array: we can leave
}



void wake_up_gpu_thread(int nbGpu){
    for( int i=0 ; i<nbGPU ; i++ ){
    DEBUG_PRT("wake up th %d",i);
    //fflush(stdout);
    sem_post(&(gpuArgs[i].sem_in));
  }

  for( int i=0 ; i<nbGPU ; i++ ){
    sem_wait(&gpuArgs[i].sem_out);
  }
}

void notify_gpus(float* progs, int* indexes, int length, CIndividual** population, int popSize, int nbGpu){

  int pop_chunk_len = popSize / nbGpu;
  //cout << " population chunk length : " << pop_chunk_len << "/" << length << endl;
  assert(nbGpu==2);
#ifdef INSTRUMENTED  
  currentStats.gpu0Blen = indexes[pop_chunk_len];
  currentStats.gpu1Blen = length-indexes[pop_chunk_len];
#endif
  sh_pop_size = pop_chunk_len;
  sh_length = length;
  
  wake_up_gpu_thread(nbGpu);
}



/**
   Send input and output data on the GPU memory.
   Allocate
*/
void initialDataToMGPU(float* input_f, int length_input, float* output_f, int length_output, int gpu_id){
  // allocate and copy input/output arrays
  CUDA_SAFE_CALL(hipMalloc((void**)(&(gpuArgs[gpu_id].inputs_k)),sizeof(float)*length_input));
  CUDA_SAFE_CALL(hipMemcpy((gpuArgs[gpu_id].inputs_k),input_f,sizeof(float)*length_input,hipMemcpyHostToDevice));

  if( output_f ){
    CUDA_SAFE_CALL(hipMalloc((void**)(&(gpuArgs[gpu_id].outputs_k)),sizeof(float)*length_output));
  }
  else {
    printf("no output buffer, dont need to allocate it\n");
    gpuArgs[gpu_id].outputs_k = NULL;
  }

  if( output_f ){
    CUDA_SAFE_CALL(hipMemcpy((gpuArgs[gpu_id].outputs_k),output_f,sizeof(float)*length_output,hipMemcpyHostToDevice));
  }
  else {
    printf("no output buffer, dont need to copy it\n");
    gpuArgs[gpu_id].outputs_k = NULL;
  }				   
  

  // allocate indexes and programs arrays
  int maxPopSize = MAX(EA->population->parentPopulationSize,EA->population->offspringPopulationSize);
  CUDA_SAFE_CALL( hipMalloc((void**)&(gpuArgs[gpu_id].indexes_k),sizeof(*indexes_k)*maxPopSize));
  CUDA_SAFE_CALL( hipMalloc((void**)&(gpuArgs[gpu_id].progs_k),sizeof(*progs_k)*MAX_PROGS_SIZE));

  // allocate hits and results arrays
  CUDA_SAFE_CALL(hipMalloc((void**)&(gpuArgs[gpu_id].results_k),sizeof(*indexes_k)*maxPopSize));
  CUDA_SAFE_CALL(hipMalloc((void**)&(gpuArgs[gpu_id].hits_k),sizeof(*indexes_k)*maxPopSize));
}


void* gpuThreadMain(void* arg){
  struct gpuArg* localArg = (struct gpuArg*)arg;

  DEBUG_PRT("gpu th %d",localArg->threadId);
  CUDA_SAFE_CALL(hipSetDevice(localArg->threadId));

  // Alloc memory for this thread
  initialDataToMGPU(inputs_f, fitnessCasesSetLength*DRONE_VAR_LEN, NULL, 0,localArg->threadId);
  
  DEBUG_PRT("allocation ok for th %d",localArg->threadId);
  //sem_post(&localArg->sem_out);

  // Wait for population to evaluate.
  while(1){
    //printf("gpu %d is evaluating\n",localArg->threadId);
    sem_wait(&localArg->sem_in);

    if( freeGPU )
      break;

    int indiv_st = localArg->threadId*sh_pop_size;
    int indiv_end = indiv_st+sh_pop_size;
    int index_st = indexes[indiv_st];
    int index_end = 0;
    if( localArg->threadId != nbGPU-1 ) index_end = indexes[indiv_end];
    else index_end = sh_length;
    
    /* cout << "gpu " << localArg->threadId << " has been notified" << endl; */
    /* cout << indiv_st << "|" << indiv_end << "|" << index_st << "|" << index_end << endl; */
    /* fflush(stdout); */

    int no_tries = 10;
    for( int i=0 ; i<no_tries ; i++ ){

      //here we copy assigned population chunk to the related GPU
      CUDA_SAFE_CALL(hipMemcpy( localArg->indexes_k, indexes+indiv_st, (indiv_end-indiv_st)*sizeof(int), hipMemcpyHostToDevice ));
      CUDA_SAFE_CALL(hipMemcpy( localArg->progs_k, progs+index_st, (index_end-index_st)*sizeof(int), hipMemcpyHostToDevice ));

#if 1
      EvaluatePostFixIndividuals_128_mgpu<<<sh_pop_size,128>>>(localArg->progs_k, index_end-index_st, sh_pop_size, localArg->inputs_k, localArg->outputs_k,
							       NB_FITNESS_CASES-1, localArg->results_k, localArg->hits_k, localArg->indexes_k, index_st, localArg->threadId);
#else
      int indivPerBlock = 4;
      dim3 numthreads;
      numthreads.x = 32;
      numthreads.y = indivPerBlock;
    
      fastEvaluatePostFixIndividuals_32_mgpu<<<sh_pop_size/indivPerBlock,numthreads,NUMTHREAD*sizeof(float)*2*indivPerBlock>>>
	(localArg->progs_k, index_end-index_st, sh_pop_size, localArg->inputs_k, localArg->outputs_k, NB_FITNESS_CASES, 
	 localArg->results_k, localArg->hits_k, indivPerBlock, localArg->indexes_k, index_st, localArg->threadId);
#endif
      /* hipDeviceSynchronize(); */
      hipError_t kernel_status = hipMemcpy( results+(localArg->threadId*sh_pop_size), localArg->results_k, (indiv_end-indiv_st)*sizeof(int), hipMemcpyDeviceToHost);
      
      if( kernel_status==hipSuccess)break;
      else cout << "try :" << i << "for generation " << EA->getCurrentGeneration() << " fails " << endl;
      
      if( i==no_tries-1){
	std::ostringstream oss;
	oss << "best-of-run-" << EA->params->seed << ".exp" ;
	EA->population->sortParentPopulation();
	ofstream fichier(oss.str().c_str(), ios::out | ios::trunc);
	fichier << EA->population->parents[0]->getFitness() << endl;
	fichier << ((IndividualImpl*)EA->population->parents[0])->hits << endl;
	fichier << treeGP_to_c( ((IndividualImpl*)EA->population->parents[0])->root[0] ) << endl;
	fichier.close();
	
	cerr << "Kernel fail to launch" << endl;
	exit(-1);
      }

    }

    CUDA_SAFE_CALL( hipMemcpy( hits+(localArg->threadId*sh_pop_size), localArg->hits_k, (indiv_end-indiv_st)*sizeof(int), hipMemcpyDeviceToHost));
    
    sem_post(&localArg->sem_out);

  }
  DEBUG_PRT("gpu : %d",localArg->threadId);
  DEBUG_PRT("addr k_prog : %p",localArg->progs_k);
  CUDA_SAFE_CALL(hipFree(localArg->progs_k));
  CUDA_SAFE_CALL(hipFree(localArg->results_k));
  CUDA_SAFE_CALL(hipFree(localArg->hits_k));
  CUDA_SAFE_CALL(hipFree(localArg->inputs_k));
  if( localArg->outputs_k ) CUDA_SAFE_CALL(hipFree(localArg->outputs_k));
  sem_post(&localArg->sem_out);
  cout << "gpu " << localArg->threadId << " has been freed" << endl;
  fflush(stdout);

  return NULL;
}
